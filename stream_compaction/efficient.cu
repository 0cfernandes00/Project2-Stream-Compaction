#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#include <iostream>

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void kernDownSweepEfficientScan(int n, int d, int* odata) {
            int k = threadIdx.x + (blockIdx.x * blockDim.x);

            int stride = 1 << (d + 1);  // 2^(d+1)

            if (k * stride >= n) return;

            int leftIdx = stride * k + (1 << d) - 1;
            int rightIdx = stride * k + stride - 1;

            int t = odata[leftIdx];

            //if (rightIdx < n) {
                odata[leftIdx] = odata[rightIdx];
                odata[rightIdx] += t;
            //}
        }


        __global__ void kernUpSweepEfficientScan(int n, int d, int* odata) {
            int index = (blockIdx.x * blockDim.x + threadIdx.x);
            int stride = 1 << (d + 1);  // 2^(d+1)

            // Only use every 2^(d+1)th thread
            if (index * stride >= n) return;

            int leftIdx = stride * index + (1 << d) - 1;   // Left child
            int rightIdx = stride * index + stride - 1;    // Right child

            if (rightIdx < n) {
                odata[rightIdx] += odata[leftIdx];
            }         
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {


            int* d_data;
            
            int paddedN = 1 << ilog2ceil(n);
            int logceilN = ilog2ceil(paddedN);

            hipMalloc((void**)&d_data, paddedN * sizeof(int));
            hipMemcpy(d_data, idata, paddedN * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
			int blockSize = 64;
			int numBlocks = (paddedN + blockSize - 1) / blockSize;


            for (int d = 0; d < logceilN; ++d) {
                int threadsNeeded = n / (1 << (d + 1));
                if (threadsNeeded == 0) break;  // No more work to do

                int numBlocksLevel = (threadsNeeded + blockSize - 1) / blockSize;


                kernUpSweepEfficientScan << <numBlocksLevel, threadsNeeded >> > (n, d, d_data);
                hipDeviceSynchronize();           
            }

            int setNMinusOne_ToZero = 0;
            hipMemcpy(d_data + paddedN - 1, &setNMinusOne_ToZero, sizeof(int), hipMemcpyHostToDevice);

            for (int d = logceilN - 1; d > -1; --d) {
                int threadsNeeded = (paddedN)/ (1 << (d + 1));
                if (threadsNeeded == 0) break;

                int numBlocksLevel = (threadsNeeded + blockSize - 1) / blockSize;

                kernDownSweepEfficientScan << <numBlocksLevel, threadsNeeded >> > (paddedN, d, d_data);
                hipDeviceSynchronize();
			}

            timer().endGpuTimer();

            hipMemcpy(odata, d_data, paddedN * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(d_data);

  
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
